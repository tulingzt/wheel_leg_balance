#include "hip/hip_runtime.h"
/**
 *  Copyright (c) 2019-2021 ETH Zurich, Automatic Control Lab,
 *  Michel Schubiger, Goran Banjac.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "cuda_pcg_interface.h"
#include "cuda_pcg.h"

#include "cuda_lin_alg.h"
#include "cuda_memory.h"

#include "glob_opts.h"
#include "profilers.h"


/*******************************************************************************
 *                           Private Functions                                 *
 *******************************************************************************/

static OSQPFloat compute_tolerance(cudapcg_solver* s,
                                   OSQPInt         admm_iter) {

  OSQPFloat eps, rhs_norm;

  /* Compute the norm of RHS of the linear system */
  cuda_vec_norm_inf(s->d_rhs, s->n, &rhs_norm);

  if (s->polishing) return c_max(rhs_norm * OSQP_CG_POLISH_TOL, OSQP_CG_TOL_MIN);

  if (admm_iter == 1) {
    // Set reduction_factor to its default value
    s->reduction_factor = s->tol_fraction;

    // In case rhs = 0.0 we don't want to set eps_prev to 0.0
    if (rhs_norm < OSQP_CG_TOL_MIN) s->eps_prev = 1.0;
    else s->eps_prev = rhs_norm * s->reduction_factor;

    // Return early since scaled_prim_res and scaled_dual_res are meaningless before the first ADMM iteration
    return s->eps_prev;
  }

  if (s->zero_pcg_iters >= s->reduction_threshold) {
    s->reduction_factor /= 2;
    s->zero_pcg_iters = 0;
  }

  eps = s->reduction_factor * sqrt((*s->scaled_prim_res) * (*s->scaled_dual_res));
  eps = c_max(c_min(eps, s->eps_prev), OSQP_CG_TOL_MIN);
  s->eps_prev = eps;

  return eps;
}

/* d_rhs = d_b1 + A' * rho * d_b2 */
static void compute_rhs(cudapcg_solver* s,
                        OSQPFloat*      d_b) {

  OSQPInt n = s->n;
  OSQPInt m = s->m;

  /* rhs = b1 */
  cuda_vec_copy_d2d(s->d_rhs, d_b, n);

  if (m == 0) return;

  /* d_z = d_b2 */
  cuda_vec_copy_d2d(s->d_z, d_b + n, m);

  if (!s->d_rho_vec) {
    /* d_z *= rho */
    cuda_vec_mult_sc(s->d_z, s->h_rho, m);
  }
  else {
    /* d_z = diag(d_rho_vec) * d_z */
    cuda_vec_ew_prod(s->d_z, s->d_z, s->d_rho_vec, m);
  }

  /* d_rhs += A' * d_z */
  cuda_mat_Axpy(s->At, s->vecz, s->vecrhs, 1.0, 1.0);
}


/*******************************************************************************
 *                              API Functions                                  *
 *******************************************************************************/

OSQPInt init_linsys_solver_cudapcg(cudapcg_solver**    sp,
                                   const OSQPMatrix*   P,
                                   const OSQPMatrix*   A,
                                   const OSQPVectorf*  rho_vec,
                                   const OSQPSettings* settings,
                                   OSQPFloat*          scaled_prim_res,
                                   OSQPFloat*          scaled_dual_res,
                                   OSQPInt             polishing) {

  OSQPInt n, m;
  OSQPFloat H_MINUS_ONE = -1.0;

  /* Allocate linsys solver structure */
  cudapcg_solver *s = (cudapcg_solver *)c_calloc(1, sizeof(cudapcg_solver));
  *sp = s;

  /* Assign type and the number of threads */
  s->type     = OSQP_INDIRECT_SOLVER;
  s->nthreads = 1;

  /* Problem dimensions */
  n = OSQPMatrix_get_n(P);
  m = OSQPMatrix_get_m(A);
  s->n = n;
  s->m = m;

  /* PCG states */
  s->polishing = polishing;
  s->zero_pcg_iters = 0;

  /* Maximum number of PCG iterations */
  s->max_iter = settings->cg_max_iter;

  /* Preconditioner to use */
  s->precond_type = settings->cg_precond;

  /* Tolerance strategy parameters */
  s->reduction_threshold = settings->cg_tol_reduction;
  s->tol_fraction        = settings->cg_tol_fraction;
  s->reduction_factor    = settings->cg_tol_fraction;
  s->scaled_prim_res     = scaled_prim_res;
  s->scaled_dual_res     = scaled_dual_res;

  /* Set pointers to problem data and ADMM settings */
  s->A            = A->S;
  s->At           = A->At;
  s->P            = P->S;
  s->d_P_diag_ind = P->d_P_diag_ind;
  s->d_rho_vec    = rho_vec ? rho_vec->d_val : NULL;

  if (polishing) {
    s->h_sigma = settings->delta;
    s->h_rho   = 1. / settings->delta;
  }
  else {
    s->h_sigma = settings->sigma;
    s->h_rho   = settings->rho;
  }

  /* Allocate raw PCG iterates */
  cuda_calloc((void **) &s->d_x,   n * sizeof(OSQPFloat));    /* Set d_x to zero */
  cuda_malloc((void **) &s->d_p,   n * sizeof(OSQPFloat));
  cuda_malloc((void **) &s->d_Kp,  n * sizeof(OSQPFloat));
  cuda_malloc((void **) &s->d_y,   n * sizeof(OSQPFloat));
  cuda_malloc((void **) &s->d_r,   n * sizeof(OSQPFloat));
  cuda_malloc((void **) &s->d_rhs, n * sizeof(OSQPFloat));
  if (m) cuda_malloc((void **) &s->d_z, m * sizeof(OSQPFloat));
  else   s->d_z = NULL;

  /* Create dense vector descriptors for PCG iterates */
  cuda_vec_create(&s->vecx,   s->d_x,   n);
  cuda_vec_create(&s->vecp,   s->d_p,   n);
  cuda_vec_create(&s->vecKp,  s->d_Kp,  n);
  cuda_vec_create(&s->vecr,   s->d_r,   n);
  cuda_vec_create(&s->vecrhs, s->d_rhs, n);
  if (m) cuda_vec_create(&s->vecz, s->d_z, m);
  else   s->vecz = NULL;

  /* Allocate scalar in host memory that is page-locked and accessible to device */
  cuda_malloc_host((void **) &s->h_r_norm, sizeof(OSQPFloat));

  /* Allocate device-side scalar values. This way scalars are packed in device memory */
  cuda_malloc((void **) &s->d_r_norm, 8 * sizeof(OSQPFloat));
  s->rTy         = s->d_r_norm + 1;
  s->rTy_prev    = s->d_r_norm + 2;
  s->alpha       = s->d_r_norm + 3;
  s->beta        = s->d_r_norm + 4;
  s->pKp         = s->d_r_norm + 5;
  s->D_MINUS_ONE = s->d_r_norm + 6;
  s->d_sigma     = s->d_r_norm + 7;
  cuda_vec_copy_h2d(s->D_MINUS_ONE, &H_MINUS_ONE, 1);
  cuda_vec_copy_h2d(s->d_sigma,     &s->h_sigma,  1);

  /* Allocate memory for PCG preconditioning */
  cuda_malloc((void **) &s->d_P_diag_val,       n * sizeof(OSQPFloat));
  cuda_malloc((void **) &s->d_AtRA_diag_val,    n * sizeof(OSQPFloat));
  cuda_malloc((void **) &s->d_diag_precond,     n * sizeof(OSQPFloat));
  cuda_malloc((void **) &s->d_diag_precond_inv, n * sizeof(OSQPFloat));
  if (!s->d_rho_vec) cuda_malloc((void **) &s->d_AtA_diag_val, n * sizeof(OSQPFloat));
  else s->d_AtA_diag_val = NULL;

  /* Link functions */
  s->name            = &name_cudapcg;
  s->solve           = &solve_linsys_cudapcg;
  s->warm_start      = &warm_start_linsys_solver_cudapcg;
  s->free            = &free_linsys_solver_cudapcg;
  s->update_matrices = &update_linsys_solver_matrices_cudapcg;
  s->update_rho_vec  = &update_linsys_solver_rho_vec_cudapcg;
  s->update_settings = &update_settings_linsys_solver_cudapcg;

  /* Initialize PCG preconditioner */
  cuda_pcg_update_precond(s, 1, 1, 1);

  /* No error */
  return 0;
}


const char* name_cudapcg(cudapcg_solver* s) {
  switch(s->precond_type) {
  case OSQP_NO_PRECONDITIONER:
    return "CUDA Conjugate Gradient - No preconditioner";
  case OSQP_DIAGONAL_PRECONDITIONER:
    return "CUDA Conjugate Gradient - Diagonal preconditioner";
  }

  return "CUDA Conjugate Gradient - Unknown preconditioner";
}


OSQPInt solve_linsys_cudapcg(cudapcg_solver* s,
                             OSQPVectorf*    b,
                             OSQPInt         admm_iter) {

  OSQPInt   pcg_iters;
  OSQPFloat eps;

  osqp_profiler_sec_push(OSQP_PROFILER_SEC_LINSYS_SOLVE);

  /* Compute the RHS of the reduced KKT system and store it in s->d_rhs */
  compute_rhs(s, b->d_val);

  /* Compute the required solution precision */
  eps = compute_tolerance(s, admm_iter);

  /* Solve the linear system with PCG */
  pcg_iters = cuda_pcg_alg(s, eps, s->max_iter);

  /* Copy the first part of the solution to b */
  cuda_vec_copy_d2d(b->d_val, s->d_x, s->n);

  if (!s->polishing) {
    /* Compute z = A * x */
    if (s->m) cuda_mat_Axpy(s->A, s->vecx, s->vecz, 1.0, 0.0);
  }
  else {
    /* Copy the second part of b to z */
    cuda_vec_copy_d2d(s->d_z, b->d_val + s->n, s->m);

    /* yred = (A * x - b2) / delta */
    cuda_mat_Axpy(s->A, s->vecx, s->vecz, 1.0, -1.0);
    cuda_vec_mult_sc(s->d_z, s->h_rho, s->m);
  }

  /* Copy the second part of the solution to b */
  if (s->m) cuda_vec_copy_d2d(b->d_val + s->n, s->d_z, s->m);

  // Number of consecutive ADMM iterations with zero PCG iterations
  if (pcg_iters == 0) s->zero_pcg_iters++;
  else                s->zero_pcg_iters = 0;

  osqp_profiler_sec_pop(OSQP_PROFILER_SEC_LINSYS_SOLVE);

  return 0;
}


void update_settings_linsys_solver_cudapcg(cudapcg_solver*     s,
                                           const OSQPSettings* settings) {

  s->max_iter            = settings->cg_max_iter;
  s->reduction_threshold = settings->cg_tol_reduction;
  s->tol_fraction        = settings->cg_tol_fraction;

  // Update preconditioner
  if (s->precond_type != settings->cg_precond) {
    s->precond_type = settings->cg_precond;

    cuda_pcg_update_precond(s, 1, 1, 1);
  }
}


void warm_start_linsys_solver_cudapcg(cudapcg_solver*    s,
                                      const OSQPVectorf* x) {

  cuda_vec_copy_d2d(s->d_x, x->d_val, x->length);
}


void free_linsys_solver_cudapcg(cudapcg_solver* s) {

  if (s) {
    /* Dense vector descriptors for PCG iterates */
    cuda_vec_destroy(s->vecx);
    cuda_vec_destroy(s->vecp);
    cuda_vec_destroy(s->vecKp);
    cuda_vec_destroy(s->vecr);
    cuda_vec_destroy(s->vecrhs);
    if (s->m) cuda_vec_destroy(s->vecz);

    /* Raw PCG iterates */
    cuda_free((void **) &s->d_x);
    cuda_free((void **) &s->d_p);
    cuda_free((void **) &s->d_Kp);
    cuda_free((void **) &s->d_y);
    cuda_free((void **) &s->d_r);
    cuda_free((void **) &s->d_rhs);
    if (s->m) cuda_free((void **) &s->d_z);

    /* Free page-locked host memory */
    cuda_free_host((void **) &s->h_r_norm);

    /* Device-side scalar values */
    cuda_free((void **) &s->d_r_norm);

    /* PCG preconditioner */
    cuda_free((void **) &s->d_P_diag_val);
    cuda_free((void **) &s->d_AtA_diag_val);
    cuda_free((void **) &s->d_AtRA_diag_val);
    cuda_free((void **) &s->d_diag_precond);
    cuda_free((void **) &s->d_diag_precond_inv);

    c_free(s);
  }
}


OSQPInt update_linsys_solver_matrices_cudapcg(cudapcg_solver*   s,
                                              const OSQPMatrix* P,
                                              const OSQPInt*    Px_new_idx,
                                              OSQPInt           P_new_n,
                                              const OSQPMatrix* A,
                                              const OSQPInt*    Ax_new_idx,
                                              OSQPInt           A_new_n) {
  /* The CUDA solver holds pointers to the matrices A and P, so it already has
     access to the updated matrices at this point. The only task remaining is to
     recompute the preconditioner */
  cuda_pcg_update_precond(s, 1, 1, 0);
  return 0;
}


OSQPInt update_linsys_solver_rho_vec_cudapcg(cudapcg_solver*    s,
                                             const OSQPVectorf* rho_vec,
                                             OSQPFloat          rho_sc) {
  /* The CUDA solver holds pointers to the rho vector, so it already has access
     to the updated vector at this point. The only task remaining is to
     recompute the preconditioner */
  s->h_rho = rho_sc;
  cuda_pcg_update_precond(s, 0, 0, 1);
  return 0;
}

