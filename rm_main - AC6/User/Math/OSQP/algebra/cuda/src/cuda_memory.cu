/**
 *  Copyright (c) 2019-2021 ETH Zurich, Automatic Control Lab,
 *  Michel Schubiger, Goran Banjac.
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "cuda_memory.h"
#include "hip/hip_runtime_api.h"    /* --> checkCudaErrors */


#define c_cudaMalloc hipMalloc
#define c_cudaMallocHost hipHostMalloc


template<typename T>
inline hipError_t  c_cudaCalloc(T** devPtr, size_t size) {
  hipError_t cudaCalloc_er = hipMalloc(devPtr, size);
  if (cudaCalloc_er == hipSuccess) {
    return hipMemset(*devPtr, 0, size);
  }
  else {
    return cudaCalloc_er;
  }
}

template<typename T>
inline hipError_t c_cudaFree(T** devPtr) {
  hipError_t cuda_error = hipFree(*devPtr);
  *devPtr = NULL;
  return cuda_error;
}

template<typename T>
inline hipError_t c_cudaFreeHost(T** devPtr) {
  hipError_t cuda_error = hipHostFree(*devPtr);
  *devPtr = NULL;
  return cuda_error;
}


void cuda_malloc(void** devPtr, size_t size) {
  checkCudaErrors(c_cudaMalloc(devPtr, size));
}

void cuda_malloc_host(void** devPtr, size_t size) {
  checkCudaErrors(c_cudaMallocHost(devPtr, size));
}

void cuda_calloc(void** devPtr, size_t size) {
  checkCudaErrors(c_cudaCalloc(devPtr, size));
}

void cuda_free(void** devPtr) {
  checkCudaErrors(c_cudaFree(devPtr));
}

void cuda_free_host(void** devPtr) {
  checkCudaErrors(c_cudaFreeHost(devPtr));
}

bool cuda_isdeviceptr(const void* ptr) {
  struct hipPointerAttribute_t attributes;

  hipError_t err = hipPointerGetAttributes(&attributes, ptr);

  // It may be possible for host allocated memory to return an error code of
  // hipErrorInvalidValue instead of hipSuccess, so we don't want to treat
  // that error code as an actual error, just as a sign the memory is on the host.
  if (err == hipErrorInvalidValue) {
    hipGetLastError();
    return false;
  }
  else if (err != hipSuccess) {
    checkCudaErrors(err);
    return false;
  }

  // This memory has been allocated on the device for sure.
  if (attributes.type == hipMemoryTypeDevice) {
    return true;
  }

  // We don't handle the cudaMemoryTypeUnregistered, hipMemoryTypeHost or hipMemoryTypeManaged
  // possibilities explicitly, because we are curious about if the memory is on the device.
  // cudaMemoryTypeUnregistered and hipMemoryTypeHost imply the memory is on the host, and
  // hipMemoryTypeManaged means the data is in memory that could be accessible from either the CPU
  // or device directly, however we always want data on the GPU (so we will do a manual copy from that
  // memory to the GPU when initializing the matrices/vectors).

  // Clear out any errors that may have happened (just in case)
  hipGetLastError();

  return false;
}

hipError_t cuda_memcpy_hd2d(void* dst, const void* src, size_t count) {
  if (cuda_isdeviceptr(src))
    return hipMemcpy(dst, src, count, hipMemcpyDeviceToDevice);
  else
    return hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
}
